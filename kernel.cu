#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include <stdio.h>
#include<stdlib.h>
#include<string.h>
#define N 3

__global__ void code(char *A, int *B, char *C, int len)
{
	int i = threadIdx.x; //cols
	int j = blockIdx.x; //rows just to see if it changes the code
	int k = blockDim.x;
	int temp = 0;
	for (int b = 0; b < len; b++)
	{
		temp = 0;
		if (j < N)
		{
			for (int c = 0; c < k; c++)
				temp = temp + ((int)A[b*k + c] * B[j*k + c]);
			if (temp != (-52))
				C[b*k + j] = (char)temp;
			else
				C[b*k + j] = '\0';
		}

	}

}

int main()
{
	// Host variables and Keys Declaration
	char str[50000], encrypt[50000], decrypt[50000];
	fgets(str, 50000, stdin);
	float etime, dtime;
	hipEvent_t estart, estop;
	int len = strlen(str);
	int ekey[N][N] = { {1,2,-1},{-2,0,1},{1,-1,0} };
	int dkey[N][N] = { {1,1,2},{1,1,1},{2,3,4} };

	// Calculating loops to process entire string	
	double a = (double)len / (double)N;
	int loop_count = ceil(a);

	// creating Events to calculate time
	hipEventCreate(&estart);
	hipEventCreate(&estop);

	// Encryption Process

		// Create Device Variables
	char *str1, *strop;
	int *key;

	// Allocate memory to Variables on Device
	hipMalloc((char**)&str1, sizeof(char)*len);
	hipMalloc((char**)&strop, sizeof(char)*len);
	hipMalloc((int**)&key, sizeof(int)*N*N);

	// Copy data from Host to Device
	hipMemcpy(str1, str, sizeof(char)*len, hipMemcpyHostToDevice);
	hipMemcpy(key, ekey, sizeof(int)*N*N, hipMemcpyHostToDevice);

	// Start event timer for encryption
	hipEventRecord(estart, 0);
	hipEventSynchronize(estart);

	//Pass it on to function and get back the cyphr
	code << <N, N >> > (str1, key, strop, loop_count);
	hipDeviceSynchronize();

	// Stop Event timmer by using another counter
	hipEventRecord(estop, 0);
	hipEventSynchronize(estop);
	hipEventElapsedTime(&etime, estart, estop);

	hipMemcpy(encrypt, strop, sizeof(char)*len, hipMemcpyDeviceToHost);

	hipFree(str1);
	hipFree(strop);
	hipFree(key);
	printf("\nEncryption : ");
	for (int i = 0; i < len; i++)
	{
		printf("%c", encrypt[i]);
	}
	printf("\n");

	// Decryption Process

		// Allocate memory to Variables on Device
	hipMalloc((char**)&str1, sizeof(char)*len);
	hipMalloc((char**)&strop, sizeof(char)*len);
	hipMalloc((int**)&key, sizeof(int)*N*N);

	// Copy data from Host to Device
	hipMemcpy(str1, encrypt, sizeof(char)*len, hipMemcpyHostToDevice);
	hipMemcpy(key, dkey, sizeof(int)*N*N, hipMemcpyHostToDevice);

	code << <N, N >> > (str1, key, strop, loop_count);
	hipDeviceSynchronize();

	hipMemcpy(decrypt, strop, sizeof(char)*len, hipMemcpyDeviceToHost);

	hipFree(str1);
	hipFree(strop);
	hipFree(key);
	printf("\n\nDecryption : ");
	for (int i = 0; i < len; i++)
	{
		printf("%c", decrypt[i]);
	}
	printf("\n");

	printf("\n\nEncryption Time : %fms\n\n", etime);
	return 0;
}
